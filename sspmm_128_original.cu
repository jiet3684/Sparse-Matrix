
#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>
#include <algorithm>
#include <stdio.h>
#include <string.h>
#include <stdlib.h>
#include <unistd.h>
//#include <cub/cub.cuh>

#define ERR fprintf(stderr, "ERR\n");

#define MIN(a,b) (((a)<(b))?(a):(b))
#define MAX(a,b) (((a)>(b))?(a):(b))
#define CEIL(a,b) (((a)+(b)-1)/(b))
#define FTYPE float
#define STYPE int

#define MFACTOR (32)
#define LOG_MFACTOR (5)
#define BSIZE (1024/1)
#define BF (BSIZE/32)
#define INIT_GRP (10000000)
#define INIT_LIST (-2147483648)
#define THRESHOLD (8*2)
#define BH (128/1)
#define BW (128/1)
#define MIN_OCC (BW)	// BW*3/4
//#define MIN_OCC (BW/4)
//#define BW (
#define SBSIZE (1024/8)
#define SBF (SBSIZE / 32)
#define DBSIZE (1024)	//1024
#define DBF (DBSIZE / 32)
#define SPBSIZE (256)
#define SPBF (SPBSIZE / 32)
#define STHRESHOLD (1024/2*1)
#define SSTRIDE (STHRESHOLD / SPBF)
#define SC_SIZE (2048)

//#define SIM_VALUE

#define GPRINT(x,y) int *tt0=(int *)malloc(sizeof(int)*(y));\
	fprintf(stderr, "\n");\
	hipMemcpy(tt0, x, sizeof(int)*(y), hipMemcpyDeviceToHost);\
	for(int i=0;i<(y);i++) if(tt0[i] == 0) fprintf(stderr,"(%d %d) ", i, tt0[i]); fprintf(stderr,"\n");\
	free(tt0);

#define GPRINT2(x,y) int *tt1=(int *)malloc(sizeof(int)*(y));\
	fprintf(stderr, "\n");\
	hipMemcpy(tt1, x, sizeof(int)*(y), hipMemcpyDeviceToHost);\
	for(int i=0;i<(y);i++) fprintf(stderr,"%d ", tt1[i]); fprintf(stderr,"\n");\
	free(tt1);


int gran=1;

struct v_struct {
	int row, col;
	FTYPE val;
	int grp;
};

double avg, vari;
struct v_struct *temp_v, *gold_temp_v;
int sc, nr, nc, ne, gold_ne, npanel, mne, mne_nr;
int nr0;

int *csr_v; 
int *csr_e0;
FTYPE *csr_ev0;


long datavol;

int compare0(const void *a, const void *b)
{
        if (((struct v_struct *)a)->row - ((struct v_struct *)b)->row > 0) return 1;
        if (((struct v_struct *)a)->row - ((struct v_struct *)b)->row < 0) return -1;
        return ((struct v_struct *)a)->col - ((struct v_struct *)b)->col;
}

void ready(int argc, char **argv)
{

        FILE *fp;
        int *loc;
        char buf[300];
        int nflag, sflag;
        int pre_count=0;
        int i;

		srand(time(NULL));
		sc = 128;
        int mode = atoi(argv[2]); // 1 : matrix-market, 2 : R-MAT
        fp = fopen(argv[1], "r");

    if (mode == 1) {
        fgets(buf, 300, fp);
        if(strstr(buf, "symmetric") != NULL || strstr(buf, "Hermitian") != NULL) sflag = 1; // symmetric
        else sflag = 0;
        if(strstr(buf, "pattern") != NULL) nflag = 0; // non-value
        else if(strstr(buf, "complex") != NULL) nflag = -1;
        else nflag = 1;

#ifdef SYM
        sflag = 1;
#endif

        while(1) {
                pre_count++;
                fgets(buf, 300, fp);
                if(strstr(buf, "%") == NULL) break;
        }
        fclose(fp);

        fp = fopen(argv[1], "r");
        for(i=0;i<pre_count;i++)
                fgets(buf, 300, fp);

        fscanf(fp, "%d %d %d", &nr, &nc, &ne);
        nr0 = nr;
        ne *= (sflag+1);

        temp_v = (struct v_struct *)malloc(sizeof(struct v_struct)*(ne+1));
        gold_temp_v = (struct v_struct *)malloc(sizeof(struct v_struct)*(ne+1));

        for(i=0;i<ne;i++) {
                fscanf(fp, "%d %d", &temp_v[i].row, &temp_v[i].col);
				temp_v[i].grp = INIT_GRP;
                temp_v[i].row--; 
                temp_v[i].col--;

                if(temp_v[i].row < 0 || temp_v[i].row >= nr || temp_v[i].col < 0 || temp_v[i].col >= nc) {
                        fprintf(stdout, "A vertex id is out of range %d %d\n", temp_v[i].row, temp_v[i].col);
                        exit(0);
                }
                if(nflag == 0) temp_v[i].val = (FTYPE)(rand()%1048576)/1048576;
                else if(nflag == 1) {
                        FTYPE ftemp;
                        fscanf(fp, " %f ", &ftemp);
                        temp_v[i].val = ftemp;
                } else { // complex
                        FTYPE ftemp1, ftemp2;
                        fscanf(fp, " %f %f ", &ftemp1, &ftemp2);
                        temp_v[i].val = ftemp1;
                }
                //temp_v[i].val = (FTYPE)(rand()%1048576)/1048576;

                if(sflag == 1) {
                        i++;
                        temp_v[i].row = temp_v[i-1].col;
                        temp_v[i].col = temp_v[i-1].row;
                        temp_v[i].val = temp_v[i-1].val;
        		temp_v[i].grp = INIT_GRP;
	        }
        }
    }
    else if (mode == 2) {

        fscanf(fp, "%d %d %d", &nc, &nr, &ne);
        nc = nr;

        temp_v = (struct v_struct *)malloc(sizeof(struct v_struct)*(ne+1));
        gold_temp_v = (struct v_struct *)malloc(sizeof(struct v_struct)*(ne+1));

        for(i=0;i<ne;i++) {
			fscanf(fp, "%d %d", &temp_v[i].row, &temp_v[i].col);
			if(temp_v[i].row == nr) temp_v[i].row--;
			if(temp_v[i].col == nr) temp_v[i].col--;
            temp_v[i].grp = INIT_GRP;
            if(temp_v[i].row < 0 || temp_v[i].row >= nr || temp_v[i].col < 0 || temp_v[i].col >= nc) {
                fprintf(stdout, "A vertex id is out of range %d %d\n", temp_v[i].row, temp_v[i].col);
                exit(0);
            }
            temp_v[i].val = (FTYPE)(rand()%1048576)/1048576;
		}
	}
	
        qsort(temp_v, ne, sizeof(struct v_struct), compare0);

        loc = (int *)malloc(sizeof(int)*(ne+1));

        memset(loc, 0, sizeof(int)*(ne+1));
        loc[0]=1;
        for(i=1;i<ne;i++) {
                if(temp_v[i].row == temp_v[i-1].row && temp_v[i].col == temp_v[i-1].col)
                        loc[i] = 0;
                else loc[i] = 1;
        }
        for(i=1;i<=ne;i++)
                loc[i] += loc[i-1];
        for(i=ne; i>=1; i--)
                loc[i] = loc[i-1];
        loc[0] = 0;

        for(i=0;i<ne;i++) {
                temp_v[loc[i]].row = temp_v[i].row;
                temp_v[loc[i]].col = temp_v[i].col;
                temp_v[loc[i]].val = temp_v[i].val;
                temp_v[loc[i]].grp = temp_v[i].grp;
        }
        ne = loc[ne];
        temp_v[ne].row = nr;
        gold_ne = ne;
        for(i=0;i<=ne;i++) {
                gold_temp_v[i].row = temp_v[i].row;
                gold_temp_v[i].col = temp_v[i].col;
                gold_temp_v[i].val = temp_v[i].val;
                gold_temp_v[i].grp = temp_v[i].grp;
        }
        free(loc);

        csr_v = (int *)malloc(sizeof(int)*(nr+1));
        csr_e0 = (int *)malloc(sizeof(int)*ne+256);
        csr_ev0 = (FTYPE *)malloc(sizeof(FTYPE)*ne+256);
        memset(csr_v, 0, sizeof(int)*(nr+1));

        for(i=0;i<ne;i++) {
                csr_e0[i] = temp_v[i].col;
                csr_ev0[i] = temp_v[i].val;
                csr_v[1+temp_v[i].row] = i+1;
        }

        for(i=1;i<nr;i++) {
                if(csr_v[i] == 0) csr_v[i] = csr_v[i-1];
        }
        csr_v[nr] = ne;

        //fprintf(stdout,"TTAAGG,%s,%d,%d,%d,",argv[1],nr0,nc,ne);
        //fprintf(fpo2,"%s,",argv[1]);

}

__global__ 
void spmv_1(int sc, int *csr_v, int *csr_e, FTYPE *csr_ev, FTYPE *vin, FTYPE *vout)
{
    int idx = (blockIdx.x*SBF)+(threadIdx.x>>5);// + (threadIdx.x>>(LOG_MFACTOR));
        int lane = (threadIdx.x&(MFACTOR-1));
        int offset = (blockIdx.z<<(LOG_MFACTOR+1))+lane;
        int offset2 = offset + MFACTOR;
        int i, j;

	FTYPE r=0.0f;
	FTYPE r2 = 0.0f;
	int loc1 = csr_v[idx], loc2 = csr_v[idx+1];

	int buf; FTYPE buf2;

	int jj = 0, l;
	for(l=loc1; l<loc2; l++) {
		if(jj == 0) {
			buf = csr_e[l+lane];
			buf2 = csr_ev[l+lane];
		}
		FTYPE v1 = __shfl(buf2, jj,MFACTOR);
		int i1 = __shfl(buf, jj,MFACTOR)*sc;
			r += v1 * vin[i1+offset];
			r2 += v1 * vin[i1+offset2];

		jj = (jj+1) % (MFACTOR-1);
	}


    
	vout[idx*sc + offset] = r;
	vout[idx*sc + offset2] = r2;
}

__global__
void spmv_2(int sc, int *csr_v, int *csr_e, FTYPE *csr_ev, FTYPE *vin, FTYPE *vout)
{
        int idx = (blockIdx.x*SBF)+(threadIdx.x>>5);// + (threadIdx.x>>(LOG_MFACTOR));
        int lane = (threadIdx.x&(MFACTOR-1));
        int offset = (blockIdx.z<<(LOG_MFACTOR+1))+lane;
        int offset2 = offset + MFACTOR;
        int i, j;

	FTYPE r=0.0f;
	FTYPE r2=0.0f;
	int loc1 = csr_v[idx], loc2 = csr_v[idx+1];

        int buf; FTYPE buf2;
        int interm3 = loc1 + (((loc2 - loc1)>>1)<<1);

    int jj=0, l;

    for(l=loc1; l<interm3; l+=2) {
		if(jj == 0) {
		        buf = csr_e[l+lane];
		        buf2 = csr_ev[l+lane];
		}
		FTYPE v1 = __shfl(buf2, jj,MFACTOR);
		FTYPE v2 = __shfl(buf2, jj+1,MFACTOR);
		int i1 = __shfl(buf, jj,MFACTOR)*sc;
		int i2 = __shfl(buf, jj+1,MFACTOR)*sc;
                r += v1 * vin[i1+offset];
                r2 += v1 * vin[i1+offset2];
                r += v2 * vin[i2+offset];
                r2 += v2 * vin[i2+offset2];

		jj = ((jj+2)&(MFACTOR-1));
        }
        if(interm3 < loc2 && jj == 0) {
                buf = csr_e[l+lane];
                buf2 = csr_ev[l+lane];
        }
        if(interm3 < loc2) {
                r += __shfl(buf2, jj,MFACTOR) * vin[__shfl(buf, jj,MFACTOR)*sc + offset];
                r2 += __shfl(buf2, jj,MFACTOR) * vin[__shfl(buf, jj,MFACTOR)*sc + offset2];
        }

	vout[idx*sc + offset] = r;
	vout[idx*sc + offset2] = r2;
}

__global__
void spmv_4(int sc, int *csr_v, int *csr_e, FTYPE *csr_ev, FTYPE *vin, FTYPE *vout)
{
	int idx = (blockIdx.x*SBF)+(threadIdx.x>>5);// + (threadIdx.x>>(LOG_MFACTOR));
	int lane = (threadIdx.x&(MFACTOR-1));
	int offset = (blockIdx.z<<(LOG_MFACTOR+1))+lane;
	int offset2 = offset + MFACTOR;
	int i, j;

FTYPE r=0.0f;
FTYPE r2=0.0f;
int loc1 = csr_v[idx], loc2 = csr_v[idx+1];

	int buf; FTYPE buf2;
	int interm2 = loc1 + (((loc2 - loc1)>>2)<<2);
	int interm3 = loc1 + (((loc2 - loc1)>>1)<<1);

int jj=0, l;	
	for(l=loc1; l<interm2; l+=4) {
			if(jj == 0) {
					buf = csr_e[l+lane]*sc;
					buf2 = csr_ev[l+lane];
			}
	FTYPE v1 = __shfl(buf2, jj,MFACTOR);
	FTYPE v2 = __shfl(buf2, jj+1,MFACTOR);
	int i1 = __shfl(buf, jj,MFACTOR);
	int i2 = __shfl(buf, jj+1,MFACTOR);
			r += v1 * vin[i1+offset];
			r2 += v1 * vin[i1+offset2];
			r += v2 * vin[i2+offset];
			r2 += v2 * vin[i2+offset2];

	FTYPE v3 = __shfl(buf2, jj+2,MFACTOR);
	FTYPE v4 = __shfl(buf2, jj+3,MFACTOR);
	int i3 = __shfl(buf, jj+2,MFACTOR);
	int i4 = __shfl(buf, jj+3,MFACTOR);
			r += v3 * vin[i3+offset];
			r2 += v3 * vin[i3+offset2];
			r += v4 * vin[i4+offset];
			r2 += v4 * vin[i4+offset2];

			jj = ((jj+4)&(MFACTOR-1));
	}
	if(interm2 < loc2 && jj == 0) {
			buf = csr_e[l+lane]*sc;
			buf2 = csr_ev[l+lane];
	}
	if(interm2 < interm3) {
	FTYPE v1 = __shfl(buf2, jj,MFACTOR);
	FTYPE v2 = __shfl(buf2, jj+1,MFACTOR);
	int i1 = __shfl(buf, jj,MFACTOR);
	int i2 = __shfl(buf, jj+1,MFACTOR);
			r += v1 * vin[i1+offset];
			r2 += v1 * vin[i1+offset2];
			r += v2 * vin[i2+offset];
			r2 += v2 * vin[i2+offset2];

			jj = (jj+2);
	}
	if(interm3 < loc2) {
			r += __shfl(buf2, jj,MFACTOR) * vin[__shfl(buf, jj,MFACTOR) + offset];
			r2 += __shfl(buf2, jj,MFACTOR) * vin[__shfl(buf, jj,MFACTOR) + offset2];
	}
vout[idx*sc + offset] = r;
vout[idx*sc + offset2] = r2;
}

__global__
void spmv_8(int sc, int *csr_v, int *csr_e, FTYPE *csr_ev, FTYPE *vin, FTYPE *vout)
{
	int idx = (blockIdx.x*SBF)+(threadIdx.x>>5);// + (threadIdx.x>>(LOG_MFACTOR));
	int lane = (threadIdx.x&(MFACTOR-1));
	int offset = (blockIdx.z<<(LOG_MFACTOR+1))+lane;
	int offset2 = offset + MFACTOR;
	int i, j;

	FTYPE r=0.0f;
	FTYPE r2=0.0f;
	int loc1 = csr_v[idx], loc2 = csr_v[idx+1];

		int buf; FTYPE buf2;
		int interm = loc1 + (((loc2 - loc1)>>3)<<3);
		int interm2 = loc1 + (((loc2 - loc1)>>2)<<2);
		int interm3 = loc1 + (((loc2 - loc1)>>1)<<1);

	int jj=0, l;	
	for(l=loc1; l<interm; l+=8) {
		if(jj == 0) {
				buf = csr_e[l+lane]*sc;
				buf2 = csr_ev[l+lane];
		}
	FTYPE v1 = __shfl(buf2, jj,MFACTOR);
	FTYPE v2 = __shfl(buf2, jj+1,MFACTOR);
	int i1 = __shfl(buf, jj,MFACTOR);
	int i2 = __shfl(buf, jj+1,MFACTOR);
		r += v1 * vin[i1+offset];
		r2 += v1 * vin[i1+offset2];
		r += v2 * vin[i2+offset];
		r2 += v2 * vin[i2+offset2];

	FTYPE v3 = __shfl(buf2, jj+2,MFACTOR);
	FTYPE v4 = __shfl(buf2, jj+3,MFACTOR);
	int i3 = __shfl(buf, jj+2,MFACTOR);
	int i4 = __shfl(buf, jj+3,MFACTOR);
		r += v3 * vin[i3+offset];
		r2 += v3 * vin[i3+offset2];
		r += v4 * vin[i4+offset];
		r2 += v4 * vin[i4+offset2];

	FTYPE v5 = __shfl(buf2, jj+4,MFACTOR);
	FTYPE v6 = __shfl(buf2, jj+5,MFACTOR);
	int i5 = __shfl(buf, jj+4,MFACTOR);
	int i6 = __shfl(buf, jj+5,MFACTOR);
		r += v5 * vin[i5+offset];
		r2 += v5 * vin[i5+offset2];
		r += v6 * vin[i6+offset];
		r2 += v6 * vin[i6+offset2];

	FTYPE v7 = __shfl(buf2, jj+6,MFACTOR);
	FTYPE v8 = __shfl(buf2, jj+7,MFACTOR);
	int i7 = __shfl(buf, jj+6,MFACTOR);
	int i8 = __shfl(buf, jj+7,MFACTOR);
		r += v7 * vin[i7+offset];
		r2 += v7 * vin[i7+offset2];
		r += v8 * vin[i8+offset];
		r2 += v8 * vin[i8+offset2];

		jj = ((jj+8)&(MFACTOR-1));
	}
	if(interm < loc2 && jj == 0) {
		buf = csr_e[l+lane]*sc;
		buf2 = csr_ev[l+lane];
	}
	if(interm < interm2) {
	FTYPE v1 = __shfl(buf2, jj,MFACTOR);
	FTYPE v2 = __shfl(buf2, jj+1,MFACTOR);
	int i1 = __shfl(buf, jj,MFACTOR);
	int i2 = __shfl(buf, jj+1,MFACTOR);
		r += v1 * vin[i1+offset];
		r2 += v1 * vin[i1+offset2];
		r += v2 * vin[i2+offset];
		r2 += v2 * vin[i2+offset2];

	FTYPE v3 = __shfl(buf2, jj+2,MFACTOR);
	FTYPE v4 = __shfl(buf2, jj+3,MFACTOR);
	int i3 = __shfl(buf, jj+2,MFACTOR);
	int i4 = __shfl(buf, jj+3,MFACTOR);
		r += v3 * vin[i3+offset];
		r2 += v3 * vin[i3+offset2];
		r += v4 * vin[i4+offset];
		r2 += v4 * vin[i4+offset2];


		jj = (jj+4);
	}
	if(interm2 < interm3) {
	FTYPE v1 = __shfl(buf2, jj,MFACTOR);
	FTYPE v2 = __shfl(buf2, jj+1,MFACTOR);
	int i1 = __shfl(buf, jj,MFACTOR);
	int i2 = __shfl(buf, jj+1,MFACTOR);
		r += v1 * vin[i1+offset];
		r2 += v1 * vin[i1+offset2];
		r += v2 * vin[i2+offset];
		r2 += v2 * vin[i2+offset2];

		jj = (jj+2);
	}
	if(interm3 < loc2) {
		r += __shfl(buf2, jj,MFACTOR) * vin[__shfl(buf, jj,MFACTOR) + offset];
		r2 += __shfl(buf2, jj,MFACTOR) * vin[__shfl(buf, jj,MFACTOR) + offset2];
	}
	vout[idx*sc + offset] = r;
	vout[idx*sc + offset2] = r2;
}

__global__
void spmv_16(int sc, int *csr_v, int *csr_e, FTYPE *csr_ev, FTYPE *vin, FTYPE *vout)
{
	int idx = (blockIdx.x*SBF)+(threadIdx.x>>5);// + (threadIdx.x>>(LOG_MFACTOR));
	int lane = (threadIdx.x&(MFACTOR-1));
	int offset = (blockIdx.z<<(LOG_MFACTOR+1))+lane;
	int offset2 = offset + MFACTOR;
	int i, j;

	FTYPE r=0.0f;
	FTYPE r2=0.0f;
	int loc1 = csr_v[idx], loc2 = csr_v[idx+1];

        int buf; FTYPE buf2;
        
		int interm0 = loc1 + (((loc2 - loc1)>>4)<<4);
		int interm = loc1 + (((loc2 - loc1)>>3)<<3);
		int interm2 = loc1 + (((loc2 - loc1)>>2)<<2);
		int interm3 = loc1 + (((loc2 - loc1)>>1)<<1);

	int jj=0, l;	
	for(l=loc1; l<interm0; l+=16) {
		if(jj == 0) {
				buf = csr_e[l+lane]*sc;
				buf2 = csr_ev[l+lane];
		}
	FTYPE v1 = __shfl(buf2, jj,MFACTOR);
	FTYPE v2 = __shfl(buf2, jj+1,MFACTOR);
	int i1 = __shfl(buf, jj,MFACTOR);
	int i2 = __shfl(buf, jj+1,MFACTOR);
		r += v1 * vin[i1+offset];
		r2 += v1 * vin[i1+offset2];
		r += v2 * vin[i2+offset];
		r2 += v2 * vin[i2+offset2];

	FTYPE v3 = __shfl(buf2, jj+2,MFACTOR);
	FTYPE v4 = __shfl(buf2, jj+3,MFACTOR);
	int i3 = __shfl(buf, jj+2,MFACTOR);
	int i4 = __shfl(buf, jj+3,MFACTOR);
		r += v3 * vin[i3+offset];
		r2 += v3 * vin[i3+offset2];
		r += v4 * vin[i4+offset];
		r2 += v4 * vin[i4+offset2];

	FTYPE v5 = __shfl(buf2, jj+4,MFACTOR);
	FTYPE v6 = __shfl(buf2, jj+5,MFACTOR);
	int i5 = __shfl(buf, jj+4,MFACTOR);
	int i6 = __shfl(buf, jj+5,MFACTOR);
		r += v5 * vin[i5+offset];
		r2 += v5 * vin[i5+offset2];
		r += v6 * vin[i6+offset];
		r2 += v6 * vin[i6+offset2];

	FTYPE v7 = __shfl(buf2, jj+6,MFACTOR);
	FTYPE v8 = __shfl(buf2, jj+7,MFACTOR);
	int i7 = __shfl(buf, jj+6,MFACTOR);
	int i8 = __shfl(buf, jj+7,MFACTOR);
		r += v7 * vin[i7+offset];
		r2 += v7 * vin[i7+offset2];
		r += v8 * vin[i8+offset];
        r2 += v8 * vin[i8+offset2];
        
    FTYPE v9 = __shfl(buf2, jj+8,MFACTOR);
	FTYPE v10 = __shfl(buf2, jj+9,MFACTOR);
	int i9 = __shfl(buf, jj+8,MFACTOR);
	int i10 = __shfl(buf, jj+9,MFACTOR);
		r += v9 * vin[i1+offset];
		r2 += v9 * vin[i1+offset2];
		r += v10 * vin[i2+offset];
		r2 += v10 * vin[i2+offset2];

	FTYPE v11 = __shfl(buf2, jj+10,MFACTOR);
	FTYPE v12 = __shfl(buf2, jj+11,MFACTOR);
	int i11 = __shfl(buf, jj+10,MFACTOR);
	int i12 = __shfl(buf, jj+11,MFACTOR);
		r += v11 * vin[i3+offset];
		r2 += v11 * vin[i3+offset2];
		r += v12 * vin[i4+offset];
		r2 += v12 * vin[i4+offset2];

	FTYPE v13 = __shfl(buf2, jj+12,MFACTOR);
	FTYPE v14 = __shfl(buf2, jj+13,MFACTOR);
	int i13 = __shfl(buf, jj+12,MFACTOR);
	int i14 = __shfl(buf, jj+13,MFACTOR);
		r += v13 * vin[i5+offset];
		r2 += v13 * vin[i5+offset2];
		r += v14 * vin[i6+offset];
		r2 += v14 * vin[i6+offset2];

	FTYPE v15 = __shfl(buf2, jj+14,MFACTOR);
	FTYPE v16 = __shfl(buf2, jj+15,MFACTOR);
	int i15 = __shfl(buf, jj+14,MFACTOR);
	int i16 = __shfl(buf, jj+15,MFACTOR);
		r += v15 * vin[i7+offset];
		r2 += v15 * vin[i7+offset2];
		r += v16 * vin[i8+offset];
		r2 += v16 * vin[i8+offset2];

		jj = ((jj+16)&(MFACTOR-1));
	}
	if(interm0 < loc2 && jj == 0) {
		buf = csr_e[l+lane]*sc;
		buf2 = csr_ev[l+lane];
    }
    if(interm0 < interm){
    FTYPE v1 = __shfl(buf2, jj,MFACTOR);
	FTYPE v2 = __shfl(buf2, jj+1,MFACTOR);
	int i1 = __shfl(buf, jj,MFACTOR);
	int i2 = __shfl(buf, jj+1,MFACTOR);
		r += v1 * vin[i1+offset];
		r2 += v1 * vin[i1+offset2];
		r += v2 * vin[i2+offset];
		r2 += v2 * vin[i2+offset2];

	FTYPE v3 = __shfl(buf2, jj+2,MFACTOR);
	FTYPE v4 = __shfl(buf2, jj+3,MFACTOR);
	int i3 = __shfl(buf, jj+2,MFACTOR);
	int i4 = __shfl(buf, jj+3,MFACTOR);
		r += v3 * vin[i3+offset];
		r2 += v3 * vin[i3+offset2];
		r += v4 * vin[i4+offset];
		r2 += v4 * vin[i4+offset2];

	FTYPE v5 = __shfl(buf2, jj+4,MFACTOR);
	FTYPE v6 = __shfl(buf2, jj+5,MFACTOR);
	int i5 = __shfl(buf, jj+4,MFACTOR);
	int i6 = __shfl(buf, jj+5,MFACTOR);
		r += v5 * vin[i5+offset];
		r2 += v5 * vin[i5+offset2];
		r += v6 * vin[i6+offset];
		r2 += v6 * vin[i6+offset2];

	FTYPE v7 = __shfl(buf2, jj+6,MFACTOR);
	FTYPE v8 = __shfl(buf2, jj+7,MFACTOR);
	int i7 = __shfl(buf, jj+6,MFACTOR);
	int i8 = __shfl(buf, jj+7,MFACTOR);
		r += v7 * vin[i7+offset];
		r2 += v7 * vin[i7+offset2];
		r += v8 * vin[i8+offset];
        r2 += v8 * vin[i8+offset2];

        jj = (jj+8);
    }
	if(interm < interm2) {
	FTYPE v1 = __shfl(buf2, jj,MFACTOR);
	FTYPE v2 = __shfl(buf2, jj+1,MFACTOR);
	int i1 = __shfl(buf, jj,MFACTOR);
	int i2 = __shfl(buf, jj+1,MFACTOR);
		r += v1 * vin[i1+offset];
		r2 += v1 * vin[i1+offset2];
		r += v2 * vin[i2+offset];
		r2 += v2 * vin[i2+offset2];

	FTYPE v3 = __shfl(buf2, jj+2,MFACTOR);
	FTYPE v4 = __shfl(buf2, jj+3,MFACTOR);
	int i3 = __shfl(buf, jj+2,MFACTOR);
	int i4 = __shfl(buf, jj+3,MFACTOR);
		r += v3 * vin[i3+offset];
		r2 += v3 * vin[i3+offset2];
		r += v4 * vin[i4+offset];
		r2 += v4 * vin[i4+offset2];


		jj = (jj+4);
	}
	if(interm2 < interm3) {
	FTYPE v1 = __shfl(buf2, jj,MFACTOR);
	FTYPE v2 = __shfl(buf2, jj+1,MFACTOR);
	int i1 = __shfl(buf, jj,MFACTOR);
	int i2 = __shfl(buf, jj+1,MFACTOR);
		r += v1 * vin[i1+offset];
		r2 += v1 * vin[i1+offset2];
		r += v2 * vin[i2+offset];
		r2 += v2 * vin[i2+offset2];

		jj = (jj+2);
	}
	if(interm3 < loc2) {
		r += __shfl(buf2, jj,MFACTOR) * vin[__shfl(buf, jj,MFACTOR) + offset];
		r2 += __shfl(buf2, jj,MFACTOR) * vin[__shfl(buf, jj,MFACTOR) + offset2];
	}
	vout[idx*sc + offset] = r;
	vout[idx*sc + offset2] = r2;
}

__global__
void spmv_32(int sc, int *csr_v, int *csr_e, FTYPE *csr_ev, FTYPE *vin, FTYPE *vout)
{
	int idx = (blockIdx.x*SBF)+(threadIdx.x>>5);// + (threadIdx.x>>(LOG_MFACTOR));
	int lane = (threadIdx.x&(MFACTOR-1));
	int offset = (blockIdx.z<<(LOG_MFACTOR+1))+lane;
	int offset2 = offset + MFACTOR;
	int i, j;

	FTYPE r=0.0f;
	FTYPE r2=0.0f;
	int loc1 = csr_v[idx], loc2 = csr_v[idx+1];

        int buf; FTYPE buf2;
        
		int interm00 = loc1 + (((loc2 - loc1)>>5)<<5);
		int interm0 = loc1 + (((loc2 - loc1)>>4)<<4);
		int interm = loc1 + (((loc2 - loc1)>>3)<<3);
		int interm2 = loc1 + (((loc2 - loc1)>>2)<<2);
		int interm3 = loc1 + (((loc2 - loc1)>>1)<<1);

	int jj=0, l;	
	for(l=loc1; l<interm00; l+=32) {
		if(jj == 0) {
				buf = csr_e[l+lane]*sc;
				buf2 = csr_ev[l+lane];
		}
	FTYPE v1 = __shfl(buf2, jj,MFACTOR);
	FTYPE v2 = __shfl(buf2, jj+1,MFACTOR);
	int i1 = __shfl(buf, jj,MFACTOR);
	int i2 = __shfl(buf, jj+1,MFACTOR);
		r += v1 * vin[i1+offset];
		r2 += v1 * vin[i1+offset2];
		r += v2 * vin[i2+offset];
		r2 += v2 * vin[i2+offset2];

	FTYPE v3 = __shfl(buf2, jj+2,MFACTOR);
	FTYPE v4 = __shfl(buf2, jj+3,MFACTOR);
	int i3 = __shfl(buf, jj+2,MFACTOR);
	int i4 = __shfl(buf, jj+3,MFACTOR);
		r += v3 * vin[i3+offset];
		r2 += v3 * vin[i3+offset2];
		r += v4 * vin[i4+offset];
		r2 += v4 * vin[i4+offset2];

	FTYPE v5 = __shfl(buf2, jj+4,MFACTOR);
	FTYPE v6 = __shfl(buf2, jj+5,MFACTOR);
	int i5 = __shfl(buf, jj+4,MFACTOR);
	int i6 = __shfl(buf, jj+5,MFACTOR);
		r += v5 * vin[i5+offset];
		r2 += v5 * vin[i5+offset2];
		r += v6 * vin[i6+offset];
		r2 += v6 * vin[i6+offset2];

	FTYPE v7 = __shfl(buf2, jj+6,MFACTOR);
	FTYPE v8 = __shfl(buf2, jj+7,MFACTOR);
	int i7 = __shfl(buf, jj+6,MFACTOR);
	int i8 = __shfl(buf, jj+7,MFACTOR);
		r += v7 * vin[i7+offset];
		r2 += v7 * vin[i7+offset2];
		r += v8 * vin[i8+offset];
        r2 += v8 * vin[i8+offset2];
        
    FTYPE v9 = __shfl(buf2, jj+8,MFACTOR);
    FTYPE v10 = __shfl(buf2, jj+9,MFACTOR);
    int i9 = __shfl(buf, jj+8,MFACTOR);
    int i10 = __shfl(buf, jj+9,MFACTOR);
        r += v9 * vin[i1+offset];
        r2 += v9 * vin[i1+offset2];
        r += v10 * vin[i2+offset];
        r2 += v10 * vin[i2+offset2];

    FTYPE v11 = __shfl(buf2, jj+10,MFACTOR);
    FTYPE v12 = __shfl(buf2, jj+11,MFACTOR);
    int i11 = __shfl(buf, jj+10,MFACTOR);
    int i12 = __shfl(buf, jj+11,MFACTOR);
        r += v11 * vin[i3+offset];
        r2 += v11 * vin[i3+offset2];
        r += v12 * vin[i4+offset];
        r2 += v12 * vin[i4+offset2];

    FTYPE v13 = __shfl(buf2, jj+12,MFACTOR);
    FTYPE v14 = __shfl(buf2, jj+13,MFACTOR);
    int i13 = __shfl(buf, jj+12,MFACTOR);
    int i14 = __shfl(buf, jj+13,MFACTOR);
        r += v13 * vin[i5+offset];
        r2 += v13 * vin[i5+offset2];
        r += v14 * vin[i6+offset];
        r2 += v14 * vin[i6+offset2];

    FTYPE v15 = __shfl(buf2, jj+14,MFACTOR);
    FTYPE v16 = __shfl(buf2, jj+15,MFACTOR);
    int i15 = __shfl(buf, jj+14,MFACTOR);
    int i16 = __shfl(buf, jj+15,MFACTOR);
        r += v15 * vin[i7+offset];
        r2 += v15 * vin[i7+offset2];
        r += v16 * vin[i8+offset];
        r2 += v16 * vin[i8+offset2];
        
    FTYPE v17 = __shfl(buf2, jj+16,MFACTOR);
	FTYPE v18 = __shfl(buf2, jj+17,MFACTOR);
	int i17 = __shfl(buf, jj+16,MFACTOR);
	int i18 = __shfl(buf, jj+17,MFACTOR);
		r += v17 * vin[i1+offset];
		r2 += v17 * vin[i1+offset2];
		r += v18 * vin[i2+offset];
		r2 += v18 * vin[i2+offset2];

	FTYPE v19 = __shfl(buf2, jj+18,MFACTOR);
	FTYPE v20 = __shfl(buf2, jj+19,MFACTOR);
	int i19 = __shfl(buf, jj+18,MFACTOR);
	int i20 = __shfl(buf, jj+19,MFACTOR);
		r += v19 * vin[i3+offset];
		r2 += v19 * vin[i3+offset2];
		r += v20 * vin[i4+offset];
		r2 += v20 * vin[i4+offset2];

	FTYPE v21 = __shfl(buf2, jj+20,MFACTOR);
	FTYPE v22 = __shfl(buf2, jj+21,MFACTOR);
	int i21 = __shfl(buf, jj+20,MFACTOR);
	int i22 = __shfl(buf, jj+21,MFACTOR);
		r += v21 * vin[i5+offset];
		r2 += v21 * vin[i5+offset2];
		r += v22 * vin[i6+offset];
		r2 += v22 * vin[i6+offset2];

	FTYPE v23 = __shfl(buf2, jj+22,MFACTOR);
	FTYPE v24 = __shfl(buf2, jj+23,MFACTOR);
	int i23 = __shfl(buf, jj+22,MFACTOR);
	int i24 = __shfl(buf, jj+23,MFACTOR);
		r += v23 * vin[i7+offset];
		r2 += v23 * vin[i7+offset2];
		r += v24 * vin[i8+offset];
        r2 += v24 * vin[i8+offset2];
        
    FTYPE v25 = __shfl(buf2, jj+24,MFACTOR);
	FTYPE v26 = __shfl(buf2, jj+25,MFACTOR);
	int i25 = __shfl(buf, jj+24,MFACTOR);
	int i26 = __shfl(buf, jj+25,MFACTOR);
		r += v25 * vin[i1+offset];
		r2 += v25 * vin[i1+offset2];
		r += v26 * vin[i2+offset];
		r2 += v26 * vin[i2+offset2];

	FTYPE v27 = __shfl(buf2, jj+26,MFACTOR);
	FTYPE v28 = __shfl(buf2, jj+27,MFACTOR);
	int i27 = __shfl(buf, jj+26,MFACTOR);
	int i28 = __shfl(buf, jj+27,MFACTOR);
		r += v27 * vin[i3+offset];
		r2 += v27 * vin[i3+offset2];
		r += v28 * vin[i4+offset];
		r2 += v28 * vin[i4+offset2];

	FTYPE v29 = __shfl(buf2, jj+28,MFACTOR);
	FTYPE v30 = __shfl(buf2, jj+29,MFACTOR);
	int i29 = __shfl(buf, jj+28,MFACTOR);
	int i30 = __shfl(buf, jj+29,MFACTOR);
		r += v29 * vin[i5+offset];
		r2 += v29 * vin[i5+offset2];
		r += v30 * vin[i6+offset];
		r2 += v30 * vin[i6+offset2];

	FTYPE v31 = __shfl(buf2, jj+30,MFACTOR);
	FTYPE v32 = __shfl(buf2, jj+31,MFACTOR);
	int i31 = __shfl(buf, jj+30,MFACTOR);
	int i32 = __shfl(buf, jj+31,MFACTOR);
		r += v31 * vin[i7+offset];
		r2 += v31 * vin[i7+offset2];
		r += v32 * vin[i8+offset];
		r2 += v32 * vin[i8+offset2];

		jj = ((jj+32)&(MFACTOR-1));
	}
	if(interm00 < loc2 && jj == 0) {
		buf = csr_e[l+lane]*sc;
		buf2 = csr_ev[l+lane];
    }
    if(interm00 < interm0){
        FTYPE v1 = __shfl(buf2, jj,MFACTOR);
	FTYPE v2 = __shfl(buf2, jj+1,MFACTOR);
	int i1 = __shfl(buf, jj,MFACTOR);
	int i2 = __shfl(buf, jj+1,MFACTOR);
		r += v1 * vin[i1+offset];
		r2 += v1 * vin[i1+offset2];
		r += v2 * vin[i2+offset];
		r2 += v2 * vin[i2+offset2];

	FTYPE v3 = __shfl(buf2, jj+2,MFACTOR);
	FTYPE v4 = __shfl(buf2, jj+3,MFACTOR);
	int i3 = __shfl(buf, jj+2,MFACTOR);
	int i4 = __shfl(buf, jj+3,MFACTOR);
		r += v3 * vin[i3+offset];
		r2 += v3 * vin[i3+offset2];
		r += v4 * vin[i4+offset];
		r2 += v4 * vin[i4+offset2];

	FTYPE v5 = __shfl(buf2, jj+4,MFACTOR);
	FTYPE v6 = __shfl(buf2, jj+5,MFACTOR);
	int i5 = __shfl(buf, jj+4,MFACTOR);
	int i6 = __shfl(buf, jj+5,MFACTOR);
		r += v5 * vin[i5+offset];
		r2 += v5 * vin[i5+offset2];
		r += v6 * vin[i6+offset];
		r2 += v6 * vin[i6+offset2];

	FTYPE v7 = __shfl(buf2, jj+6,MFACTOR);
	FTYPE v8 = __shfl(buf2, jj+7,MFACTOR);
	int i7 = __shfl(buf, jj+6,MFACTOR);
	int i8 = __shfl(buf, jj+7,MFACTOR);
		r += v7 * vin[i7+offset];
		r2 += v7 * vin[i7+offset2];
		r += v8 * vin[i8+offset];
        r2 += v8 * vin[i8+offset2];
        
    FTYPE v9 = __shfl(buf2, jj+8,MFACTOR);
    FTYPE v10 = __shfl(buf2, jj+9,MFACTOR);
    int i9 = __shfl(buf, jj+8,MFACTOR);
    int i10 = __shfl(buf, jj+9,MFACTOR);
        r += v9 * vin[i1+offset];
        r2 += v9 * vin[i1+offset2];
        r += v10 * vin[i2+offset];
        r2 += v10 * vin[i2+offset2];

    FTYPE v11 = __shfl(buf2, jj+10,MFACTOR);
    FTYPE v12 = __shfl(buf2, jj+11,MFACTOR);
    int i11 = __shfl(buf, jj+10,MFACTOR);
    int i12 = __shfl(buf, jj+11,MFACTOR);
        r += v11 * vin[i3+offset];
        r2 += v11 * vin[i3+offset2];
        r += v12 * vin[i4+offset];
        r2 += v12 * vin[i4+offset2];

    FTYPE v13 = __shfl(buf2, jj+12,MFACTOR);
    FTYPE v14 = __shfl(buf2, jj+13,MFACTOR);
    int i13 = __shfl(buf, jj+12,MFACTOR);
    int i14 = __shfl(buf, jj+13,MFACTOR);
        r += v13 * vin[i5+offset];
        r2 += v13 * vin[i5+offset2];
        r += v14 * vin[i6+offset];
        r2 += v14 * vin[i6+offset2];

    FTYPE v15 = __shfl(buf2, jj+14,MFACTOR);
    FTYPE v16 = __shfl(buf2, jj+15,MFACTOR);
    int i15 = __shfl(buf, jj+14,MFACTOR);
    int i16 = __shfl(buf, jj+15,MFACTOR);
        r += v15 * vin[i7+offset];
        r2 += v15 * vin[i7+offset2];
        r += v16 * vin[i8+offset];
        r2 += v16 * vin[i8+offset2];

        jj = (jj+16);
    }
    if(interm0 < interm){
    FTYPE v1 = __shfl(buf2, jj,MFACTOR);
	FTYPE v2 = __shfl(buf2, jj+1,MFACTOR);
	int i1 = __shfl(buf, jj,MFACTOR);
	int i2 = __shfl(buf, jj+1,MFACTOR);
		r += v1 * vin[i1+offset];
		r2 += v1 * vin[i1+offset2];
		r += v2 * vin[i2+offset];
		r2 += v2 * vin[i2+offset2];

	FTYPE v3 = __shfl(buf2, jj+2,MFACTOR);
	FTYPE v4 = __shfl(buf2, jj+3,MFACTOR);
	int i3 = __shfl(buf, jj+2,MFACTOR);
	int i4 = __shfl(buf, jj+3,MFACTOR);
		r += v3 * vin[i3+offset];
		r2 += v3 * vin[i3+offset2];
		r += v4 * vin[i4+offset];
		r2 += v4 * vin[i4+offset2];

	FTYPE v5 = __shfl(buf2, jj+4,MFACTOR);
	FTYPE v6 = __shfl(buf2, jj+5,MFACTOR);
	int i5 = __shfl(buf, jj+4,MFACTOR);
	int i6 = __shfl(buf, jj+5,MFACTOR);
		r += v5 * vin[i5+offset];
		r2 += v5 * vin[i5+offset2];
		r += v6 * vin[i6+offset];
		r2 += v6 * vin[i6+offset2];

	FTYPE v7 = __shfl(buf2, jj+6,MFACTOR);
	FTYPE v8 = __shfl(buf2, jj+7,MFACTOR);
	int i7 = __shfl(buf, jj+6,MFACTOR);
	int i8 = __shfl(buf, jj+7,MFACTOR);
		r += v7 * vin[i7+offset];
		r2 += v7 * vin[i7+offset2];
		r += v8 * vin[i8+offset];
        r2 += v8 * vin[i8+offset2];

        jj = (jj+8);
    }
	if(interm < interm2) {
	FTYPE v1 = __shfl(buf2, jj,MFACTOR);
	FTYPE v2 = __shfl(buf2, jj+1,MFACTOR);
	int i1 = __shfl(buf, jj,MFACTOR);
	int i2 = __shfl(buf, jj+1,MFACTOR);
		r += v1 * vin[i1+offset];
		r2 += v1 * vin[i1+offset2];
		r += v2 * vin[i2+offset];
		r2 += v2 * vin[i2+offset2];

	FTYPE v3 = __shfl(buf2, jj+2,MFACTOR);
	FTYPE v4 = __shfl(buf2, jj+3,MFACTOR);
	int i3 = __shfl(buf, jj+2,MFACTOR);
	int i4 = __shfl(buf, jj+3,MFACTOR);
		r += v3 * vin[i3+offset];
		r2 += v3 * vin[i3+offset2];
		r += v4 * vin[i4+offset];
		r2 += v4 * vin[i4+offset2];


		jj = (jj+4);
	}
	if(interm2 < interm3) {
	FTYPE v1 = __shfl(buf2, jj,MFACTOR);
	FTYPE v2 = __shfl(buf2, jj+1,MFACTOR);
	int i1 = __shfl(buf, jj,MFACTOR);
	int i2 = __shfl(buf, jj+1,MFACTOR);
		r += v1 * vin[i1+offset];
		r2 += v1 * vin[i1+offset2];
		r += v2 * vin[i2+offset];
		r2 += v2 * vin[i2+offset2];

		jj = (jj+2);
	}
	if(interm3 < loc2) {
		r += __shfl(buf2, jj,MFACTOR) * vin[__shfl(buf, jj,MFACTOR) + offset];
		r2 += __shfl(buf2, jj,MFACTOR) * vin[__shfl(buf, jj,MFACTOR) + offset2];
	}
	vout[idx*sc + offset] = r;
	vout[idx*sc + offset2] = r2;
}

void process()
{

	int i, j;

	int *_csr_v; int *_csr_e0; FTYPE *_csr_ev0;
	int *_csr_e; FTYPE *_csr_ev;


        hipMalloc((void **) &_csr_v, sizeof(int)*(nr+1));
        hipMalloc((void **) &_csr_e0, sizeof(int)*ne+256);
        hipMalloc((void **) &_csr_ev0, sizeof(FTYPE)*ne+256);

        hipMemset(_csr_v, 0, sizeof(int)*(nr+1));
        hipMemset(_csr_e0, 0, sizeof(int)*ne+256);
        hipMemset(_csr_ev0, 0, sizeof(FTYPE)*ne+256);

        hipMemcpy(_csr_v, csr_v, sizeof(int)*(nr+1), hipMemcpyHostToDevice);
        hipMemcpy(_csr_e0, csr_e0, sizeof(int)*(ne+1), hipMemcpyHostToDevice);
        hipMemcpy(_csr_ev0, csr_ev0, sizeof(FTYPE)*ne, hipMemcpyHostToDevice);


        FTYPE *vin, *_vin, *vout, *_vout;
        FTYPE *vout_gold;
        vin = (FTYPE *)malloc(sizeof(FTYPE)*nc*sc);
        vout = (FTYPE *)malloc(sizeof(FTYPE)*nr*sc);
        vout_gold = (FTYPE *)malloc(sizeof(FTYPE)*nr*sc);

        hipError_t err = hipSuccess;

        err = hipMalloc((void **) &_vin, sizeof(FTYPE)*nc*sc);
        if(err != 0) exit(0);
        err = hipMalloc((void **) &_vout, sizeof(FTYPE)*nr*sc);
        if(err != 0) exit(0);

        hipMemset(_vout, 0, sizeof(FTYPE)*nr*sc);
        for(i=0;i<nc*sc;i++) {
                vin[i] = (FTYPE)(rand()%1048576)/1048576;
#ifdef SIM_VALUE
		vin[i] = 1;
#endif
        }
		hipMemcpy(_vin, vin, sizeof(FTYPE)*nc*sc, hipMemcpyHostToDevice);
		
        hipStream_t stream1, stream2, stream3;
        hipStreamCreate(&stream1);
        hipStreamCreate(&stream2);
		hipStreamCreate(&stream3);
		
	// process
	dim3 s_gridsize(nr/SBF, 1, CEIL(sc, MFACTOR*2));
	dim3 s_blocksize(SBSIZE, 1, 1);
	dim3 ss_gridsize(nr, 1, 1);
	dim3 ss_blocksize(SBSIZE, 1, 1);


	float tot_ms;
	hipEvent_t event1, event2;
	hipEventCreate(&event1);
	hipEventCreate(&event2);

    hipDeviceSynchronize();
	hipEventRecord(event1,0);
    spmv_1<<<s_gridsize, s_blocksize, 0, stream1>>>(sc, _csr_v, _csr_e0, _csr_ev0, _vin, _vout);
	hipEventRecord(event2,0);
	hipEventSynchronize(event1);
	hipEventSynchronize(event2);
	hipEventElapsedTime(&tot_ms, event1, event2);

	hipDeviceSynchronize();
	hipEventRecord(event1,0);
    spmv_2<<<s_gridsize, s_blocksize, 0, stream1>>>(sc, _csr_v, _csr_e0, _csr_ev0, _vin, _vout);
	hipEventRecord(event2,0);
	hipEventSynchronize(event1);
	hipEventSynchronize(event2);
	hipEventElapsedTime(&tot_ms, event1, event2);

	hipDeviceSynchronize();
	hipEventRecord(event1,0);
    spmv_4<<<s_gridsize, s_blocksize, 0, stream1>>>(sc, _csr_v, _csr_e0, _csr_ev0, _vin, _vout);
	hipEventRecord(event2,0);
	hipEventSynchronize(event1);
	hipEventSynchronize(event2);
	hipEventElapsedTime(&tot_ms, event1, event2);

	hipDeviceSynchronize();
	hipEventRecord(event1,0);
    spmv_8<<<s_gridsize, s_blocksize, 0, stream1>>>(sc, _csr_v, _csr_e0, _csr_ev0, _vin, _vout);
	hipEventRecord(event2,0);
	hipEventSynchronize(event1);
	hipEventSynchronize(event2);
    hipEventElapsedTime(&tot_ms, event1, event2);
    
    hipDeviceSynchronize();
	hipEventRecord(event1,0);
    spmv_16<<<s_gridsize, s_blocksize, 0, stream1>>>(sc, _csr_v, _csr_e0, _csr_ev0, _vin, _vout);
	hipEventRecord(event2,0);
	hipEventSynchronize(event1);
	hipEventSynchronize(event2);
    hipEventElapsedTime(&tot_ms, event1, event2);
    
    hipDeviceSynchronize();
	hipEventRecord(event1,0);
    spmv_32<<<s_gridsize, s_blocksize, 0, stream1>>>(sc, _csr_v, _csr_e0, _csr_ev0, _vin, _vout);
	hipEventRecord(event2,0);
	hipEventSynchronize(event1);
	hipEventSynchronize(event2);
	hipEventElapsedTime(&tot_ms, event1, event2);

//	fprintf(stdout, "\n");
        free(vin); free(vout); hipFree(_vin); hipFree(_vout);
        free(vout_gold);
	printf("\n");
//printf("st ; %d\n", SSTRIDE);
}

int main(int argc, char **argv)
{
	ready(argc, argv);
	//gen_structure();
	process();
}

